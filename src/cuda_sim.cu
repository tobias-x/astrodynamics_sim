
#include <hip/hip_runtime.h>
#ifdef USE_CUDA
#include "cuda_sim.h"
#include <cuda.h>
#include <cuda_runtime.h>
#include <vector>
#include <iostream>

static constexpr double G_CONST = 6.6743e-11;
static constexpr double SOFTENING = 1e-5;

__global__ void newtonianKernel(Body* bodies, int n, double dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    extern __shared__ Body shared[];
    for (int j=threadIdx.x; j<n; j+=blockDim.x)
        shared[j] = bodies[j];
    __syncthreads();

    Body me = shared[i];
    bodies[i] = me;
}

void CudaSim::step(std::vector<Body>& bodies, double dt) const {
    int n = bodies.size();
    Body* d_bodies;
    cudaMalloc(&d_bodies, n*sizeof(Body));
    cudaMemcpy(d_bodies, bodies.data(), n*sizeof(Body), cudaMemcpyHostToDevice);

    int block = 128, grid = (n+block-1)/block;
    newtonianKernel<<<grid,block,n*sizeof(Body)>>>(d_bodies, n, dt);
    cudaDeviceSynchronize();

    cudaMemcpy(bodies.data(), d_bodies, n*sizeof(Body), cudaMemcpyDeviceToHost);
    cudaFree(d_bodies);
}

void CudaSim::runSim(const std::vector<Body>& bodies,
                     double timeStep,
                     int totalSteps,
                     std::vector<SimulationResult>& results) const
{
    std::vector<Body> state = bodies;
    for (int step=0; step<totalSteps; ++step) {
        step(state, timeStep);
        for (int i=0; i<(int)state.size(); ++i) {
            results.push_back({
                step, i,
                state[i].x, state[i].y, state[i].z,
                state[i].vx, state[i].vy, state[i].vz
            });
        }
    }
}
#endif
